// Author: Ulises Olivares
// uolivares@unam.mx
// June 8, 2023


#include "common.h"
#include "Methods_IDW_CD.h"
#include "Raster.h"
#include <ctime>
#include <chrono>
#include <string.h>
#include <tclap/CmdLine.h>
#include <omp.h>
#include <sstream>
#include <iostream>
#include <fstream>
#include <unordered_map>


// structure definition
typedef struct locality{
    int row;
    int col;
    int ID;
    float demand;
}locality;

typedef struct localities{
    int year;
    locality * locsArray;
}localities;


// Methods definition
void parseParameters(int argc, const char** argv);
void RunCDIDW(string frictionMap, string demmandFile, string locsMap, string scenario);
float* importRaster(string name, int &rows, int &cols, float &scale, int &cell_null);
float* importLocsRaster(std::string name, int &rows, int &cols, float &scale, int &cell_null, long long int &countLocs, std::map<int, std::pair<int, int>> &matrixMap);
void loadCSVDemmand(string name, float *locsMatrix, unordered_map<int, vector<float>> &demand);
int generateLocsStruct(float *map_local, int rows, int cols, localities &locs, int cell_null, unordered_map<int, vector<float>> &demand, std::map<int, std::pair<int, int>> &matrixMap);
float* resetMatrix(int rows,  int cols, float val1);

//Global variable definition
string frictionMapW, demmandFileW, locsMapW, frictionMapV, demmandFileV, locsMapV;
int isFriccRelative = 1, numThreads = 0, hours = 12;
float exponent = 1.005;

// Raster variables
const char *projection;
double adfGeoTransform[6];


int main(int argc, const char** argv) {
    // Parse parameters from command line
    parseParameters(argc, argv);

    // Run CDIDW for walking Scenario
    RunCDIDW(frictionMapW, demmandFileW, locsMapW, "walking");

    return 0;
}

/*
 * This function start the CD and IDW algorithm for a given scenario.
 * Input: str <friction file tiff>, str <demmand file csv>
 */
void RunCDIDW(string frictionMap, string demmandFile, string locsMap, string scenario){
    //Object creation
    //Raster objrast;

    // Variable declaration
    int rows, cols, nullValue = 0, locsNum= 0;
    float scale;// map scale
    unordered_map<int, vector<float>> demand;// Vector to store demand of all years
    //map<int, locality> localities;// map of localities
    localities *locs;
    const int moves[2][8]={{1, 1, 0, -1, -1, -1, 0,  1},{0, 1, 1, 1,  0,  -1, -1, -1}}; // all possible combinations of movements in a map including diagonals


    //Import friction matrix
    float* fric_matrix, *locsMatrix, *IDW_matrix; //matrices to store input/output data

    // Import friction raster
    fric_matrix = importRaster(frictionMap, rows, cols, scale, nullValue);

    // Initializar IDWMatrix with 0's
    IDW_matrix = resetMatrix(rows, cols, 0);

    // Import Localities map
    long long int numLocs=0;
    // TODO: store locs ands its position into a struct
    std::map<int, std::pair<int, int>> matrixMap;
    locsMatrix = importLocsRaster(locsMap, rows, cols, scale, nullValue, numLocs, matrixMap);

    //cout << "number of locs in map: " << matrixMap.size() << endl;
    //cout << "Number of locs in  tiff: " << numLocs << endl;

    cout <<"Loading demand..." << endl;
    // Load demand per year
    loadCSVDemmand(demmandFile, locsMatrix, demand);


    // count the number of localities
    locsNum = generateLocsStruct(locsMatrix, rows, cols, locs, nullValue, demand, matrixMap);
    cout << "Total number of localities " <<" " << locsNum << endl;


    int sizeLocs = locsNum * sizeof(localities);

    cout << locsNum <<endl;

    for (int i = 0; i < locsNum; i++) {
        localities* currentLocs = &locs[i];
        int year = currentLocs->year;
        locality* locsArray = currentLocs->locsArray;

        // Iterate over the localities within the current localities struct
        for (int j = 0; j < locsNum; j++) {
            locality* currentLocality = &locsArray[j];
            // Access and work with the current locality
            int row = currentLocality->row;
            int col = currentLocality->col;
            int ID = currentLocality->ID;
            float demand = currentLocality->demand;

            cout << ID << "=>" << demand << endl;

            // ... do something with row, col, ID, and demand
        }
    }
    // Biomass requirement
    /*map<int, float> requiredBiomass;
    for(int year= 1; year < demand.size(); year++){
        for(int loc=0; loc < demand[0].second.size(); loc++){ // Tamaño de localidades
            requiredBiomass.insert(pair<int, float>(int(demand[0].second[loc]), float(demand[year].second[loc])));//load demand in tons
        }
    }*/

    // 1) Declare host variables
    float* d_fric_matrix, * d_locsMatrix, *d_IDW_matrix;

    size_t  size = rows * cols *sizeof(float);


    // 2) Allocate device memory
    hipError_t cudaStatus = hipMalloc((void**)&d_fric_matrix, size);
    cudaStatus = hipMalloc((void**)&d_locsMatrix, size);
    cudaStatus = hipMalloc((void**)&d_IDW_matrix, size);
    //hipError_t cudaStatus = hipMalloc((void**)&d_IDW_matrix, size);

    //TODO: validate each device memory allocation
    if (cudaStatus != hipSuccess) {
        cout<< "Error in cuda memory allocation: " << cudaStatus << endl;
    }

    // 3) Copy data from host  to devide IDW, fric, locs
    hipMemcpy(d_fric_matrix, fric_matrix, size, hipMemcpyHostToDevice);
    hipMemcpy(d_fric_matrix, d_locsMatrix, size, hipMemcpyHostToDevice); // Not necessary
    hipMemcpy(d_fric_matrix, d_IDW_matrix, size, hipMemcpyHostToDevice);


    // 4) Instantiate the Kernel

    // 5) Copy memory from device

    // 6) free cuda mem

    hipFree(d_fric_matrix);
    hipFree(d_locsMatrix);
    hipFree(d_IDW_matrix);

    // TODO: start the parallel calculation of CD
}

/*
 * This function initializes a matrix with a specific value.
 * Input: Rows, cols and value
 * Output: Matrix with the givens size and value
 */
float* resetMatrix(int rows, int cols, float val1){
    int row,col;
    float *IDW = new float [rows * cols];//matriz IDW para todas las localidades
    for(row=0; row<rows; row++)
        for(col=0; col<cols; col++)
            IDW[(cols*row)+col]=val1;
    return(IDW);
}

/*
 * This function builds a struct of localities with all data
 */
int generateLocsStruct(float *map_local, int rows, int cols, localities &locs, int cell_null, unordered_map<int, vector<float>> &demand, std::map<int, std::pair<int, int>> &matrixMap) {
    int countLoc = matrixMap.size(); // Size of locs stored in the CSV
    //locs = (localities*)malloc(demand.size() * sizeof(localities));
    int size = matrixMap.size() - 2;
    locs = new localities[size];
    for (int year = 0; year < demand.size(); year++) {
        locs[year].year = year;
        locs[year].locsArray = new locality[size];
        int locTmp = 0;

        for (const auto &entry: matrixMap) {
            locs[year].locsArray[locTmp].ID = entry.first;
            locs[year].locsArray[locTmp].row = entry.second.first;
            locs[year].locsArray[locTmp].col = entry.second.second;
            auto it = demand.find(entry.first);
            locs[year].locsArray[locTmp].demand = it->second[year];
            locTmp++;
        }
    }
    return countLoc;
}


/*
 * This function reads localities from a CSV file and stores their demmand per year.
 * Input: CSV filename
 * Output: a vector with ID and demmand per year.
 */
void loadCSVDemmand(string name, float *locsMatrix, unordered_map<int, vector<float>> &demand){
    //localities * demand = new ;
    //vector<pair<string, vector<float>>> result;
    //unordered_map<int, vector<float>> demand;
    // Create an input filestream
    std::ifstream myFile(name);
    // Make sure the file is open
    if(!myFile.is_open())
        throw runtime_error("Could not open file");
    // Helper variables
    string line, colname, value;
   // float val;
    // Step 1) Read the column names ignore this information
    if(myFile.good())
    {
        // Extract the first line in the file
        getline(myFile, line);

        // Create a stringstream from line
        stringstream ss(line);

        // Extract each column name // This while are goint to finish once all titles have completed
        while(getline(ss, colname, ',')){
            colname.erase(remove(colname.begin(), colname.end(), '"'), colname.end());
            // Initialize and add <colname, int vector> pairs to result
            //result.push_back({colname, vector<float> {}});
            //demand[colname] = std::vector<float>{};
        }
    }
    else
    {
        cout << "An error occurred while reading the CSV file!" << endl;
        exit(0);
    }

    // Step 2) Read data, row by rows
    while(getline(myFile, line))
    {
        // Create a stringstream of the current line
        stringstream ss(line);
        // Keep track of the current column index
        int colIdx = 0, id = 0;
        while(getline(ss, value, ',')){
            //cout << value << endl;
            // Convert first value to numeric
            if(colIdx == 0){
                value.erase(remove(value.begin(), value.end(), '"'), value.end()); // remove special " char.
                id = stoi(value);
                demand[id] = std::vector<float>{};// TODO:
            }
            else
                demand[id].push_back(stof(value));// Remove values only for the first element

            // If the next token is a comma, ignore it and move on
            if(ss.peek() == ',') ss.ignore();
            // Increment the column index
            colIdx++;

        }
    }
    // Close file
    myFile.close();

    //return result;
}

float* importRaster(std::string name, int &rows, int &cols, float &scale, int &cell_null){
    int row,col;//iteradores matriz
    GDALDataset *dataset;
    GDALAllRegister();
    string ds = name;
    dataset = (GDALDataset *) GDALOpen(ds.c_str(), GA_ReadOnly);
    GDALRasterBand  *poBand;

    poBand = dataset->GetRasterBand(1);
    dataset->GetGeoTransform( adfGeoTransform );

    projection = dataset->GetProjectionRef();

    cols = poBand->GetXSize();
    rows= poBand->GetYSize();
    scale = adfGeoTransform[1];

    dataset->GetGeoTransform( adfGeoTransform );
    scale = adfGeoTransform[1];
    cell_null = poBand->GetNoDataValue();  //read null value of cell
    float *matrix = new float[rows * cols];
    float *pBuf = new float[rows * cols];

    if (poBand->RasterIO(GF_Read, 0, 0, cols, rows, pBuf, cols, rows, GDT_Float32, 0, 0) == 0)
        cout << name << " raster imported sucessfully!" << endl;
    else
        cout << "An error occurred during the importation process! " << endl;

    int location;
    for (row = 0; row < rows; row++)
        for ( col = 0; col < cols; col++){
            location = (cols * (row)) + col;
            matrix[(cols*row)+col] = *(pBuf+location);
        }
    //cout<<"valor nulo: "<< cell_null<< endl;
    return matrix;

}

float* importLocsRaster(std::string name, int &rows, int &cols, float &scale, int &cell_null, long long int &countLocs, std::map<int, std::pair<int, int>> &matrixMap){
    int row,col;//iteradores matriz
    //std::map<int, std::pair<int, int>> matrixMap; // map to store
    GDALDataset *dataset;
    GDALAllRegister();
    string ds = name;
    dataset = (GDALDataset *) GDALOpen(ds.c_str(), GA_ReadOnly);
    GDALRasterBand  *poBand;

    poBand = dataset->GetRasterBand(1);
    dataset->GetGeoTransform( adfGeoTransform );

    projection = dataset->GetProjectionRef();

    cols = poBand->GetXSize();
    rows= poBand->GetYSize();
    scale = adfGeoTransform[1];

    dataset->GetGeoTransform( adfGeoTransform );
    scale = adfGeoTransform[1];
    cell_null = poBand->GetNoDataValue();  //read null value of cell
    float *matrix = new float[rows * cols];
    float *pBuf = new float[rows * cols];

    if (poBand->RasterIO(GF_Read, 0, 0, cols, rows, pBuf, cols, rows, GDT_Float32, 0, 0) == 0)
        cout << name << " raster imported sucessfully!" << endl;
    else
        cout << "An error occurred during the importation process! " << endl;

    int location;
    for (row = 0; row < rows; row++)
        for ( col = 0; col < cols; col++){
            location = (cols * (row)) + col;
            matrix[(cols*row)+col] = *(pBuf+location);
            if (matrix[(cols*row)+col] != cell_null && matrix[(cols*row)+col] != 0){
                matrixMap[matrix[(cols*row)+col]] = std::make_pair(row, col);
                countLocs++;
            }

        }
    return matrix;
}


void parseParameters(int argc, const char** argv){
    try {
        // Define the command line object, and insert a message
        // that describes the program. The "Command description message"
        // is printed last in the help text. The second argument is the
        // delimiter (usually space) and the last one is the version number.
        // The CmdLine object parses the argv array based on the Arg objects
        // that it contains.
        TCLAP::CmdLine cmd("Command description message", ' ', "1");

        // Parse the argv array.
        //cmd.parse( argc, argv );


        // Define a value argument and add it to the command line.
        // A value arg defines a flag and a type of value that it expects,
        // such as "-n Bishop".
        //TCLAP::ValueArg<std::string> nameArg("n","name","Name to print",true,"homer","string");
        TCLAP::ValueArg<std::string>frictionw("1","frictionWalking","Absolute path to friction.tif for walking scenario",true,"/path/to/friction.tif","string");
        TCLAP::ValueArg<std::string>locsw("2","locsWalking","Absolute path to locs.tif for walking scenario",true,"/path/to/locs.tif","string");
        TCLAP::ValueArg<std::string>demmandw("3","demmandWalking","Absolute path to demmand.csv for walking scenario",true,"/path/to/demmand.csv","string");

        TCLAP::ValueArg<std::string>frictionv("4","frictionVehicle","Absolute path to friction.tif for vehicle scenario",true,"/path/to/friction.tif","string");
        TCLAP::ValueArg<std::string>locsv("5","locsVehicle","Absolutse path to locs.tif for vehicle scenario",true,"/path/to/locs.tif","string");
        TCLAP::ValueArg<std::string>demmandv("6","demmandVehicle","Absolute path to demmand.csv for vehicle scenario",true,"/path/to/demmand.csv","string");

        TCLAP::ValueArg<std::string>relative("r","relative","1 to friction relative, 0 otherwise",true,"1 to friction relative, 0 otherwise","string");
        TCLAP::ValueArg<std::string>processors("p","processors","Number of processors -p <int>",true,"Number of processors -p <int>","string");
        TCLAP::ValueArg<std::string>timeLimit("t","timeLimit","Time limit for Cost Distance exploration (hours) -t <int>",true,"Time limit for exploration -t <int>","string");
        TCLAP::ValueArg<std::string>expo("e","exponent","IDW exponent -e <float>",true,"IDW exponent -e <float>","string");
        // Add the argument nameArg to the CmdLine object. The CmdLine object
        // uses this Arg to parse the command line.


        cmd.add(frictionw);
        cmd.add(locsw);
        cmd.add(demmandw);

        cmd.add(frictionv);
        cmd.add(locsv);
        cmd.add(demmandv);

        cmd.add(relative);
        cmd.add(processors);
        cmd.add(timeLimit);
        cmd.add(expo);


        // Parse the argv array.
        cmd.parse( argc, argv );

        // Walking
        frictionMapW = frictionw.getValue();
        demmandFileW = demmandw.getValue();
        locsMapW = locsw.getValue();

        // Vehicle
        frictionMapV = frictionv.getValue();
        demmandFileV = demmandv.getValue();
        locsMapV = locsv.getValue();

        // user defined
        isFriccRelative = stoi(relative.getValue());
        numThreads = stoi(processors.getValue());
        hours = stoi(timeLimit.getValue());
        exponent = stof(expo.getValue());

    }
    catch (TCLAP::ArgException &e)  // catch any exceptions
    {
        std::cerr << "error: " << e.error() << " for arg " << e.argId() << std::endl;
        exit(0);
    }
}