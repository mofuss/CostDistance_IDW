// Author: Ulises Olivares
// uolivares@unam.mx
// June 8, 2023


#include "common.h"
#include "Methods_IDW_CD.h"
#include "Raster.h"
#include <ctime>
#include <chrono>
#include <string.h>
#include <tclap/CmdLine.h>
#include <omp.h>
#include <sstream>
#include <iostream>
#include <fstream>


// structure definition
typedef struct locality{
    int row;
    int col;
    int ID;
    float demand;
}locality;

typedef struct localities{
    int year;
    locality * locs;
}localities;

// Methods definition
void parseParameters(int argc, const char** argv);
void RunCDIDW(string frictionMap, string demmandFile, string locsMap, string scenario);
float* importRaster(string name, int &rows, int &cols, float &scale, int &cell_null);
float* importLocsRaster(std::string name, int &rows, int &cols, float &scale, int &cell_null, long long int &countLocs);
vector<pair<string, vector<float>>> loadDemmand(string name, float *locsMatrix);
int readLocalities(float *map_local, int rows, int cols, map<int,locality> &local_ord, int cell_null, vector<pair<string, vector<float>>> demand);
float* resetMatrix(int rows,  int cols, float val1);

//Global variable definition
string frictionMapW, demmandFileW, locsMapW, frictionMapV, demmandFileV, locsMapV;
int isFriccRelative = 1, numThreads = 0, hours = 12;
float exponent = 1.005;

// Raster variables
const char *projection;
double adfGeoTransform[6];


int main(int argc, const char** argv) {
    // Parse parameters from command line
    parseParameters(argc, argv);

    // Run CDIDW for walking Scenario
    RunCDIDW(frictionMapW, demmandFileW, locsMapW, "walking");

    return 0;
}

/*
 * This function start the CD and IDW algorithm for a given scenario.
 * Input: str <friction file tiff>, str <demmand file csv>
 */
void RunCDIDW(string frictionMap, string demmandFile, string locsMap, string scenario){
    //Object creation
    //Raster objrast;

    // Variable declaration
    int rows, cols, nullValue = 0, locsNum= 0;
    float scale;// map scale
    vector<pair<string, vector<float>>> demand;// Vector to store demand of all years
    map<int, locality> localities;// map of localities
    const int moves[2][8]={{1, 1, 0, -1, -1, -1, 0,  1},{0, 1, 1, 1,  0,  -1, -1, -1}}; // all possible combinations of movements in a map including diagonals


    //Import friction matrix
    float* fric_matrix, *locsMatrix, *IDW_matrix; //matrices to store input/output data

    // Import friction raster
    fric_matrix = importRaster(frictionMap, rows, cols, scale, nullValue);

    // Initializar IDWMatrix with 0's
    IDW_matrix = resetMatrix(rows, cols, 0);

    // Import Localities map
    long long int locs=0;
    locsMatrix = importLocsRaster(locsMap, rows, cols, scale, nullValue, locs);

    //cout <<"Loading demand..." << endl;
    // Load demand per year
    demand = loadDemmand(demmandFile, locsMatrix);


    // count the number of localities
    locsNum = readLocalities(locsMatrix, rows, cols, localities, nullValue, demand);
    cout << "Total number of localities " << locs <<" " << locsNum << endl;

    // Biomass requirement
    /*map<int, float> requiredBiomass;
    for(int year= 1; year < demand.size(); year++){
        for(int loc=0; loc < demand[0].second.size(); loc++){ // Tamaño de localidades
            requiredBiomass.insert(pair<int, float>(int(demand[0].second[loc]), float(demand[year].second[loc])));//load demand in tons
        }
    }*/

    // 1) Declare host variables
    float* d_fric_matrix, * d_locsMatrix, *d_IDW_matrix;

    size_t  size = rows * cols *sizeof(float);


    // 2) Allocate device memory
    hipError_t cudaStatus = hipMalloc((void**)&d_fric_matrix, size);
    cudaStatus = hipMalloc((void**)&d_locsMatrix, size);
    cudaStatus = hipMalloc((void**)&d_IDW_matrix, size);
    //hipError_t cudaStatus = hipMalloc((void**)&d_IDW_matrix, size);

    //TODO: validate each device memory allocation
    if (cudaStatus != hipSuccess) {
        cout<< "Error in cuda memory allocation: " << cudaStatus << endl;
    }

    // 3) Copy data from host  to devide IDW, fric, locs
    hipMemcpy(d_fric_matrix, fric_matrix, size, hipMemcpyHostToDevice);
    hipMemcpy(d_fric_matrix, d_locsMatrix, size, hipMemcpyHostToDevice); // Not necessary
    hipMemcpy(d_fric_matrix, d_IDW_matrix, size, hipMemcpyHostToDevice);


    // 4) Instantiate the Kernel

    // 5) Copy memory from device

    // 6) free cuda mem

    hipFree(d_fric_matrix);
    hipFree(d_locsMatrix);
    hipFree(d_IDW_matrix);

    // TODO: start the parallel calculation of CD
}

/*
 * This function initializes a matrix with a specific value.
 * Input: Rows, cols and value
 * Output: Matrix with the givens size and value
 */
float* resetMatrix(int rows, int cols, float val1){
    int row,col;
    float *IDW = new float [rows * cols];//matriz IDW para todas las localidades
    for(row=0; row<rows; row++)
        for(col=0; col<cols; col++)
            IDW[(cols*row)+col]=val1;
    return(IDW);
}

/*
 * This function counts the number of localities in a map
 * TODO: this might be unnecesary in future releases.
 */
int readLocalities(float *map_local, int rows, int cols, map<int,locality> &local_ord, int cell_null, vector<pair<string, vector<float>>> demand) {
    //cout << "Enter to readLocs" << endl;
    locality array;
    int countLoc = 0;

    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            if (map_local[(cols * row) + col] != cell_null) {
                //if (countLoc == 1)
                //    break;
                array.row = row;
                array.col = col;
                //cout << "raster ID :" << map_local[(cols * row) + col] <<endl;

                int rasterID = int(map_local[(cols * row) + col]); //rasterized map

                for (int year = 1;year < demand.size();year++){
                    for(int loc=0; loc < demand[0].second.size(); loc++){ // Tamaño de localidades
                        array.ID =  int(demand[0].second[loc]);
                        //cout << "Id: " << id <<" ";
                        array.demand = float(demand[year].second[loc]);//load demand in tons
                        //cout << "Demand: " << d << endl;
                    }
                }
                //array.demand = ;//add demmand
                // modify this
                local_ord[(int) map_local[(cols * row) + col]] = array;
                countLoc++;
            }
        }
    }
    return countLoc;
}
/*
 * This function reads localities from a CSV file and stores their demmand per year.
 * Input: CSV filename
 * Output: a vector with ID and demmand per year.
 */

vector<pair<string, vector<float>>> loadDemmand(string name, float *locsMatrix){
    //localities * result2 = new ;
    vector<pair<string, vector<float>>> result;
    // Create an input filestream
    std::ifstream myFile(name);

    // Make sure the file is open
    if(!myFile.is_open())
        throw runtime_error("Could not open file");


    // Helper variables
    string line, colname, value;
   // float val;

    // Step 1) Read the column names
    if(myFile.good())
    {
        // Extract the first line in the file
        getline(myFile, line);

        // Create a stringstream from line
        stringstream ss(line);

        // Extract each column name // This while are goint to finish once all titles have completed
        while(getline(ss, colname, ',')){

            colname.erase(remove(colname.begin(), colname.end(), '"'), colname.end());
            // Initialize and add <colname, int vector> pairs to result
            result.push_back({colname, vector<float> {}});
        }
    }
    else
    {
        cout << "An error occurred while reading the CSV file!" << endl;
        exit(0);
    }

    // Step 2) Read data, row by rows
    while(getline(myFile, line))
    {
        // Create a stringstream of the current line
        stringstream ss(line);

        // Keep track of the current column index
        int colIdx = 0, id;
        while(getline(ss, value, ',')){
            //cout << value << endl;
            // Convert first value to numeric
            if(colIdx == 0){
                value.erase(remove(value.begin(), value.end(), '"'), value.end()); // remove special " char.
                id = stoi(value);
            } // Remove values only for the first element


            // TODO: complete into a single structure all data:
            // ID, x, y, demand, year
            //result2[colIdx]

            // Add value to vector
            result.at(colIdx).second.push_back(stof(value));

            // If the next token is a comma, ignore it and move on
            if(ss.peek() == ',') ss.ignore();
            // Increment the column index
            colIdx++;

        }
    }

    // Close file
    myFile.close();

    return result;
}

float* importRaster(std::string name, int &rows, int &cols, float &scale, int &cell_null){
    int row,col;//iteradores matriz
    GDALDataset *dataset;
    GDALAllRegister();
    string ds = name;
    dataset = (GDALDataset *) GDALOpen(ds.c_str(), GA_ReadOnly);
    GDALRasterBand  *poBand;

    poBand = dataset->GetRasterBand(1);
    dataset->GetGeoTransform( adfGeoTransform );

    projection = dataset->GetProjectionRef();

    cols = poBand->GetXSize();
    rows= poBand->GetYSize();
    scale = adfGeoTransform[1];

    dataset->GetGeoTransform( adfGeoTransform );
    scale = adfGeoTransform[1];
    cell_null = poBand->GetNoDataValue();  //read null value of cell
    float *matrix = new float[rows * cols];
    float *pBuf = new float[rows * cols];

    if (poBand->RasterIO(GF_Read, 0, 0, cols, rows, pBuf, cols, rows, GDT_Float32, 0, 0) == 0)
        cout << name << " raster imported sucessfully!" << endl;
    else
        cout << "An error occurred during the importation process! " << endl;

    int location;
    for (row = 0; row < rows; row++)
        for ( col = 0; col < cols; col++){
            location = (cols * (row)) + col;
            matrix[(cols*row)+col] = *(pBuf+location);
        }
    //cout<<"valor nulo: "<< cell_null<< endl;
    return matrix;

}

float* importLocsRaster(std::string name, int &rows, int &cols, float &scale, int &cell_null, long long int &countLocs){
    int row,col;//iteradores matriz
    GDALDataset *dataset;
    GDALAllRegister();
    string ds = name;
    dataset = (GDALDataset *) GDALOpen(ds.c_str(), GA_ReadOnly);
    GDALRasterBand  *poBand;

    poBand = dataset->GetRasterBand(1);
    dataset->GetGeoTransform( adfGeoTransform );

    projection = dataset->GetProjectionRef();

    cols = poBand->GetXSize();
    rows= poBand->GetYSize();
    scale = adfGeoTransform[1];

    dataset->GetGeoTransform( adfGeoTransform );
    scale = adfGeoTransform[1];
    cell_null = poBand->GetNoDataValue();  //read null value of cell
    float *matrix = new float[rows * cols];
    float *pBuf = new float[rows * cols];

    if (poBand->RasterIO(GF_Read, 0, 0, cols, rows, pBuf, cols, rows, GDT_Float32, 0, 0) == 0)
        cout << name << " raster imported sucessfully!" << endl;
    else
        cout << "An error occurred during the importation process! " << endl;

    int location;
    for (row = 0; row < rows; row++)
        for ( col = 0; col < cols; col++){
            location = (cols * (row)) + col;
            matrix[(cols*row)+col] = *(pBuf+location);

            if (matrix[(cols*row)+col] != cell_null && matrix[(cols*row)+col] != 0){
                //cout << "mat: " <<matrix[(cols*row)+col]<< endl;
                countLocs++;
            }

        }
    //cout<<"Total locs: "<< countLocs<< endl;
    return matrix;

}


void parseParameters(int argc, const char** argv){
    try {
        // Define the command line object, and insert a message
        // that describes the program. The "Command description message"
        // is printed last in the help text. The second argument is the
        // delimiter (usually space) and the last one is the version number.
        // The CmdLine object parses the argv array based on the Arg objects
        // that it contains.
        TCLAP::CmdLine cmd("Command description message", ' ', "1");

        // Parse the argv array.
        //cmd.parse( argc, argv );


        // Define a value argument and add it to the command line.
        // A value arg defines a flag and a type of value that it expects,
        // such as "-n Bishop".
        //TCLAP::ValueArg<std::string> nameArg("n","name","Name to print",true,"homer","string");
        TCLAP::ValueArg<std::string>frictionw("1","frictionWalking","Absolute path to friction.tif for walking scenario",true,"/path/to/friction.tif","string");
        TCLAP::ValueArg<std::string>locsw("2","locsWalking","Absolute path to locs.tif for walking scenario",true,"/path/to/locs.tif","string");
        TCLAP::ValueArg<std::string>demmandw("3","demmandWalking","Absolute path to demmand.csv for walking scenario",true,"/path/to/demmand.csv","string");

        TCLAP::ValueArg<std::string>frictionv("4","frictionVehicle","Absolute path to friction.tif for vehicle scenario",true,"/path/to/friction.tif","string");
        TCLAP::ValueArg<std::string>locsv("5","locsVehicle","Absolute path to locs.tif for vehicle scenario",true,"/path/to/locs.tif","string");
        TCLAP::ValueArg<std::string>demmandv("6","demmandVehicle","Absolute path to demmand.csv for vehicle scenario",true,"/path/to/demmand.csv","string");

        TCLAP::ValueArg<std::string>relative("r","relative","1 to friction relative, 0 otherwise",true,"1 to friction relative, 0 otherwise","string");
        TCLAP::ValueArg<std::string>processors("p","processors","Number of processors -p <int>",true,"Number of processors -p <int>","string");
        TCLAP::ValueArg<std::string>timeLimit("t","timeLimit","Time limit for Cost Distance exploration (hours) -t <int>",true,"Time limit for exploration -t <int>","string");
        TCLAP::ValueArg<std::string>expo("e","exponent","IDW exponent -e <float>",true,"IDW exponent -e <float>","string");
        // Add the argument nameArg to the CmdLine object. The CmdLine object
        // uses this Arg to parse the command line.


        cmd.add(frictionw);
        cmd.add(locsw);
        cmd.add(demmandw);

        cmd.add(frictionv);
        cmd.add(locsv);
        cmd.add(demmandv);

        cmd.add(relative);
        cmd.add(processors);
        cmd.add(timeLimit);
        cmd.add(expo);


        // Parse the argv array.
        cmd.parse( argc, argv );

        // Walking
        frictionMapW = frictionw.getValue();
        demmandFileW = demmandw.getValue();
        locsMapW = locsw.getValue();

        // Vehicle
        frictionMapV = frictionv.getValue();
        demmandFileV = demmandv.getValue();
        locsMapV = locsv.getValue();

        // user defined
        isFriccRelative = stoi(relative.getValue());
        numThreads = stoi(processors.getValue());
        hours = stoi(timeLimit.getValue());
        exponent = stof(expo.getValue());

    }
    catch (TCLAP::ArgException &e)  // catch any exceptions
    {
        std::cerr << "error: " << e.error() << " for arg " << e.argId() << std::endl;
        exit(0);
    }
}